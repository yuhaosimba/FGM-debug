#include "hip/hip_runtime.h"
#include "Potential-Calculator.cuh"

void Charge_Interpolation(){
    refresh_to_zero_potential << <(nx * ny * nz + 255) / 256, 256 >> > (d_charge_ortho, nx * ny * nz);
    charge_interpolation << <(n_charge + 255) / 256, 256 >> > (d_charge_crd, d_charge_discrete, nx, ny, nz, box_x, box_y, box_z, dx, dy, dz, n_charge, d_charge_ortho);
    trans_from_ortho_to_calc << <(nx * ny * nz + 255) / 256, 256 >> > (d_charge_ortho, d_charge, d_from_ortho_to_calc, nx, ny, nz);
    hipblasDaxpy(bandle, A_num_rows, &one, d_const, 1, d_charge, 1);
    printf("charge-interpolation finished\n");
}

void Potential_file_saver() {
    hipMemcpy(result_phi, d_phi, A_num_rows * sizeof(double), hipMemcpyDeviceToHost);
    // clean result.txt
    std::ofstream f_clean("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\result.txt");
    // write result to txt
    std::ofstream f_result("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\result.txt");
    for (int i = 0; i < A_num_rows; i++) { f_result << result_phi[i] << std::endl; }
}

void Safe_cuda_free() {
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vec_phi);
    hipsparseDestroyDnVec(vec_r);
    hipsparseDestroyDnVec(vec_p);
    hipsparseDestroyDnVec(vec_Ap);
    hipblasDestroy(bandle);
    hipsparseDestroy(handle);

    hipFree(dBuffer);
    hipFree(dA_csrOffsets); hipFree(dA_columns); hipFree(dA_values);
    hipFree(d_phi); hipFree(d_const);
    hipFree(d_r); hipFree(d_p); hipFree(d_Ap);
    hipFree(d_charge); hipFree(d_charge_ortho);
    hipFree(d_charge_crd); hipFree(d_charge_discrete);
}



void CG_Solver(hipsparseHandle_t handle, double minus, hipsparseSpMatDescr_t matA, hipsparseDnVecDescr_t vec_phi,
    double zero, hipsparseDnVecDescr_t vec_r, hipblasHandle_t bandle, int A_num_rows, double one, double* d_const, double* d_r,
    double* d_p, double err, double r_r, hipsparseDnVecDescr_t vec_p, hipsparseDnVecDescr_t vec_Ap, double* d_Ap, double p_A_p,
    double alpha, double* d_phi, double r_r_next, double beta, void* dBuffer) {

    // Initialize r = b - Ax
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus, matA, vec_phi, &zero, vec_r, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
    hipblasDaxpy(bandle, A_num_rows, &one, d_const, 1, d_r, 1);
    hipMemcpy(d_p, d_r, A_num_rows * sizeof(double), hipMemcpyDeviceToDevice);

    // Initialize err
    hipblasDnrm2(bandle, A_num_rows, d_r, 1, &err);

    // CG-iteration
    while (err >= 1e-5) {
        // alpha = (r, r) / (Ap, p)
        hipblasDdot(bandle, A_num_rows, d_r, 1, d_r, 1, &r_r);
        hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, vec_p, &zero, vec_Ap, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
        hipblasDdot(bandle, A_num_rows, d_Ap, 1, d_p, 1, &p_A_p);
        alpha = r_r / p_A_p;

        // phi = phi + alpha * p;  r = r - alpha * Ap
        hipblasDaxpy(bandle, A_num_rows, &alpha, d_p, 1, d_phi, 1);
        alpha = -alpha;
        hipblasDaxpy(bandle, A_num_rows, &alpha, d_Ap, 1, d_r, 1);

        // calc new (r,r) ;  beta = (r,r)_new / (r,r)
        hipblasDdot(bandle, A_num_rows, d_r, 1, d_r, 1, &r_r_next);
        beta = r_r_next / r_r;

        // calc new p = r + beta * p
        hipblasDscal(bandle, A_num_rows, &beta, d_p, 1);
        hipblasDaxpy(bandle, A_num_rows, &one, d_r, 1, d_p, 1);

        // calc err
        hipblasDnrm2(bandle, A_num_rows, d_r, 1, &err);
        printf("err = %0.9f\n", err);
    }
}



// Debug ���� Crd �� ���Դ洢���� i �� ��ɵ� x ����Ϊ crd[3*i], y ����Ϊ crd[3*i+1], z ����Ϊ crd[3*i+2]
__global__ void charge_interpolation(double* crd, double* charge_q, int nx, int ny, int nz, double box_x, double box_y, double box_z,
    double dx, double dy, double dz, int n, double* d_charge_ortho) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_x, idx_y, idx_z; // ������ڵ�������������
    int idx_x_plus, idx_y_plus, idx_z_plus; // ������ڵ����������������һ��,����3d�����Ա߽�����
    double xx, yy, zz; // ����ڸ����������еĹ�һ��λ��
    if (i < n) {
        idx_x = (crd[3 * i] / dx);
        idx_y = (crd[3 * i + 1] / dy);
        idx_z = (crd[3 * i + 2] / dz);
        xx = (crd[3 * i] - idx_x * dx) / dx;
        yy = (crd[3 * i + 1] - idx_y * dy) / dy;
        zz = (crd[3 * i + 2] - idx_z * dz) / dz;
        idx_x_plus = (idx_x + 1) % nx;
        idx_y_plus = (idx_y + 1) % ny;
        idx_z_plus = (idx_z + 1) % nz;
        double q = charge_q[i];
        // α���������Բ�ֵ
        d_charge_ortho[idx_x + idx_y * nx + idx_z * nx * ny] += (1 - xx) * (1 - yy) * (1 - zz) * q;
        d_charge_ortho[idx_x_plus + idx_y * nx + idx_z * nx * ny] += xx * (1 - yy) * (1 - zz) * q;
        d_charge_ortho[idx_x + idx_y_plus * nx + idx_z * nx * ny] += (1 - xx) * yy * (1 - zz) * q;
        d_charge_ortho[idx_x_plus + idx_y_plus * nx + idx_z * nx * ny] += xx * yy * (1 - zz) * q;
        d_charge_ortho[idx_x + idx_y * nx + idx_z_plus * nx * ny] += (1 - xx) * (1 - yy) * zz * q;
        d_charge_ortho[idx_x_plus + idx_y * nx + idx_z_plus * nx * ny] += xx * (1 - yy) * zz * q;
        d_charge_ortho[idx_x + idx_y_plus * nx + idx_z_plus * nx * ny] += (1 - xx) * yy * zz * q;
        d_charge_ortho[idx_x_plus + idx_y_plus * nx + idx_z_plus * nx * ny] += xx * yy * zz * q;
    }
    return;
}

__global__ void refresh_to_zero_potential(double* d_phi_ortho, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        d_phi_ortho[i] = 0;
    }
}

__global__ void trans_from_ortho_to_calc(double* d_phi_ortho, double* d_phi, int* d_from_ortho_to_calc, int nx, int ny, int nz) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nx * ny * nz) {
        if (d_from_ortho_to_calc[i] != -1) {
            d_phi[d_from_ortho_to_calc[i]] = d_phi_ortho[i];
        }
    }
}

__global__ void trans_from_calc_to_ortho(double* d_phi, double* d_phi_ortho, int* d_from_calc_to_ortho, int first_cut_sgn) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < first_cut_sgn) {
        d_phi_ortho[d_from_calc_to_ortho[i]] = d_phi[i];
    }
}

void Mesh_initialize() {
    std::ifstream f_info("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\info.txt");
    f_info >> A_num_rows >> A_nnz >> nx >> ny >> nz >> box_x >> box_y >> box_z >> first_cut_sgn;
    f_info.close();
    dx = box_x / nx;
    dy = box_y / ny;
    dz = box_z / nz;

    // ����Ԫ���񡢳�ʼ���ơ�������ɳ�������
    hA_csrOffsets = (int*)malloc((A_num_rows + 1) * sizeof(int));
    hA_columns = (int*)malloc(A_nnz * sizeof(int));
    hA_values = (double*)malloc(A_nnz * sizeof(double));
    h_phi = (double*)malloc(A_num_rows * sizeof(double));
    h_const = (double*)malloc(A_num_rows * sizeof(double));

    // �Ӽ����������������ӳ��
    h_from_calc_to_ortho = (int*)malloc(first_cut_sgn * sizeof(int));
    h_from_ortho_to_calc = (int*)malloc(nx * ny * nz * sizeof(int));

    // ��������任�����������������
    h_phi_ortho = (double*)malloc(nx * ny * nz * sizeof(double));   // λ����(i,j,k)�ϵ��������Ϊ i + j*NX + k*NX*NY




    // ���������
    result_phi = (double*)malloc(nx * ny * nz * sizeof(double));

    // Read parameters to host
    std::ifstream f_indptr("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\indptr.txt");
    for (int i = 0; i < A_num_rows + 1; i++) { f_indptr >> hA_csrOffsets[i]; }
    f_indptr.close();
    std::ifstream f_indices("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\indices.txt");
    for (int i = 0; i < A_nnz; i++) { f_indices >> hA_columns[i]; }
    f_indices.close();
    std::ifstream f_data("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\data.txt");
    for (int i = 0; i < A_nnz; i++) { f_data >> hA_values[i]; }
    f_data.close();

    std::ifstream f_phi("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\phi.txt");
    for (int i = 0; i < A_num_rows; i++) { f_phi >> h_phi[i]; }
    f_phi.close();
    std::ifstream f_charge("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\Const.txt");
    for (int i = 0; i < A_num_rows; i++) { f_charge >> h_const[i]; }
    f_charge.close();

    std::ifstream f_trans("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\calc_need_convert_list.txt");
    for (int i = 0; i < first_cut_sgn; i++) { f_trans >> h_from_calc_to_ortho[i]; }
    f_trans.close();

    std::ifstream f_trans2("C:\\Users\\15653\\Desktop\\FGM-Python-Debug\\CSR-Matrix\\ortho_need_convert_list.txt");
    for (int i = 0; i < nx * ny * nz; i++) { f_trans2 >> h_from_ortho_to_calc[i]; }
    f_trans2.close();


    hipMalloc((void**)&dA_csrOffsets, (A_num_rows + 1) * sizeof(int));
    hipMalloc((void**)&dA_columns, A_nnz * sizeof(int));
    hipMalloc((void**)&dA_values, A_nnz * sizeof(double));
    hipMalloc((void**)&d_phi, A_num_rows * sizeof(double));
    hipMalloc((void**)&d_const, A_num_rows * sizeof(double));
    hipMalloc((void**)&d_phi_ortho, nx * ny * nz * sizeof(double));  // �����������


    hipMalloc((void**)&d_charge, A_num_rows * sizeof(double)); // ������
    hipMalloc((void**)&d_charge_ortho, (nx * ny * nz) * sizeof(double)); // ����������

    hipMalloc((void**)&d_r, A_num_rows * sizeof(double));
    hipMalloc((void**)&d_p, A_num_rows * sizeof(double));
    hipMalloc((void**)&d_Ap, A_num_rows * sizeof(double));

    hipMalloc((void**)&d_charge_crd, 3 * n_charge * sizeof(double));
    hipMalloc((void**)&d_charge_discrete, n_charge * sizeof(double));

    hipMalloc((void**)&d_from_calc_to_ortho, first_cut_sgn * sizeof(int));
    hipMalloc((void**)&d_from_ortho_to_calc, nx * ny * nz * sizeof(int));

    // Copy data from host to device
    hipMemcpy(dA_csrOffsets, hA_csrOffsets, (A_num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_values, hA_values, A_nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_phi, h_phi, A_num_rows * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_const, h_const, A_num_rows * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_charge_crd, h_charge_crd, n_charge * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_charge_discrete, h_charge_discrete, n_charge * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_from_calc_to_ortho, h_from_calc_to_ortho, first_cut_sgn * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_from_ortho_to_calc, h_from_ortho_to_calc, nx * ny * nz * sizeof(int), hipMemcpyHostToDevice);

    finish_t = clock();
    total_t = (double)(finish_t - start_t) / CLOCKS_PER_SEC;


    // CUSPARSE & CUBLAS APIs 
    hipblasCreate(&bandle);
    hipsparseCreate(&handle);

    // Create matrix and vector descriptors
    hipsparseCreateCsr(&matA, A_num_rows, A_num_rows, A_nnz, dA_csrOffsets, dA_columns, dA_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    hipsparseCreateDnVec(&vec_phi, A_num_rows, d_phi, HIP_R_64F);
    hipsparseCreateDnVec(&vec_phi_ortho, nx * ny * nz, d_phi_ortho, HIP_R_64F);
    hipsparseCreateDnVec(&vec_r, A_num_rows, d_r, HIP_R_64F);
    hipsparseCreateDnVec(&vec_p, A_num_rows, d_p, HIP_R_64F);
    hipsparseCreateDnVec(&vec_Ap, A_num_rows, d_Ap, HIP_R_64F);
    hipsparseCreateDnVec(&vec_charge, A_num_rows, d_charge, HIP_R_64F);
    hipsparseCreateDnVec(&vec_charge_ortho, nx * ny * nz, d_charge_ortho, HIP_R_64F);
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minus, matA, vec_phi, &zero, vec_r, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    printf("Initialization & memory copy host -> device \n");
    printf("Time used = %f\n", total_t);
}